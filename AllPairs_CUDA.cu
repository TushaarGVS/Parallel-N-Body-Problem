
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <omp.h>
#include <time.h>
#include <sys/time.h>
#define THREADS_PER_BLOCK 2048


struct vec{
  float x, y ;
} ;

struct particle{
  float x, y, m ;
} ;

__global__ void find_force(int index, struct particle *particles, struct vec *ans, int size){
  
  struct particle p = particles[index] ;
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  
  if(i <  size && !(particles[i].x == p.x && particles[i].y == p.y)){ 
    float dsq = pow(p.x - particles[i].x, 2) ;
    dsq += pow(p.y - particles[i].y, 2) ;
    float d = sqrt(dsq) ;

    __syncthreads() ;
    ans[index].x += ((particles[i].x - p.x) * ( particles[i].m / pow(d,3))) ;
    ans[index].y += ((particles[i].y - p.y) * ( particles[i].m / pow(d,3))) ;
    __syncthreads() ;
  }
}

double get_time() {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

int main(){
  int n ;
  double start, end ;
  scanf("%d" ,&n) ;

  struct particle particles[n] ;

  float r,vx,vy, cr,cg,cb ;
  scanf("%f" ,&r) ;
  for(int i= 0 ; i < n ; i++)
    scanf("%f%f%f%f%f%f%f%f", &particles[i].x, &particles[i].y, &vx, &vy, &particles[i].m, &cr, &cg, &cb) ;
  

  struct vec ans[n] ;

  struct particle *d_particles ;
  struct vec *d_ans ;
  int d_particles_size = n * sizeof(struct particle) ;
  int d_ans_size = n * sizeof(struct vec) ;
    
  hipMalloc((void **)&d_particles, d_particles_size) ;
  hipMalloc((void **)&d_ans, d_ans_size) ;

  hipMemcpy(d_particles, particles, d_particles_size, hipMemcpyHostToDevice) ;
	    
  start = get_time() ;

  for(int i = 0 ; i < n ; i++)
    find_force<<<(n+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(i, d_particles, d_ans, n) ;
    

  hipMemcpy(ans, d_ans, d_ans_size, hipMemcpyDeviceToHost) ;

  end = get_time() ;

  printf("%lf\n", (double)(end-start)) ;
}
